
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialConvolutionMM.cu"
#else

static inline void THNN_(SpatialConvolutionMM_shapeCheck)(
                         THCState *state,
                         THCTensor *input, THCTensor *gradOutput,
                         THCTensor *weight, THCTensor *bias,
                         int kH, int kW, int dH, int dW, int padH, int padW,
                         int weight_nullable) {
  THArgCheck(kW > 0 && kH > 0, 9,
             "kernel size should be greater than zero, but got kH: %d kW: %d", kH, kW);
  THArgCheck(dW > 0 && dH > 0, 11,
             "stride should be greater than zero, but got dH: %d dW: %d", dH, dW);

  if (weight != NULL) {
    THCUNN_argCheck(state, !weight->is_empty() && (weight->dim() == 2 || weight->dim() == 4), 5, weight,
                    "non-empty 2D or 4D weight tensor expected, but got: %s");
    if (bias != NULL) {
      THCUNN_check_dim_size(state, bias, 1, 0, weight->size(0));
    }
  } else if (!weight_nullable) {
    THError("weight tensor is expected to be non-nullable");
  }

  int ndim = input->dim();
  int dimf = 0;
  int dimh = 1;
  int dimw = 2;

  if (ndim == 4) {
    dimf++;
    dimh++;
    dimw++;
  }

  THCUNN_argCheck(state, !input->is_empty() && (ndim == 3 || ndim == 4), 2, input,
                  "non-empty 3D or 4D input tensor expected but got: %s");

  int64_t inputHeight  = input->size(dimh);
  int64_t inputWidth   = input->size(dimw);

  int64_t exactInputHeight = inputHeight + 2 * padH;
  int64_t exactInputWidth = inputWidth + 2 * padW;

  if (exactInputHeight < kH || exactInputWidth < kW) {
    THError("Calculated padded input size per channel: (%ld x %ld). "
      "Kernel size: (%ld x %ld). Kernel size can't be greater than actual input size",
      exactInputHeight, exactInputWidth, kH, kW);
  }

  int64_t outputHeight = (exactInputHeight - kH) / dH + 1;
  int64_t outputWidth  = (exactInputWidth - kW) / dW + 1;

  if (outputWidth < 1 || outputHeight < 1) {
    THError("Given input size per channel: (%ld x %ld). "
      "Calculated output size per channel: (%ld x %ld). Output size is too small",
      inputHeight, inputWidth, outputHeight, outputWidth);
  }

  if (weight != NULL) {
    int64_t nInputPlane = weight->size(1);
    if (weight->dim() == 2) {
      nInputPlane /= (kH * kW);
    }
    THCUNN_check_dim_size(state, input, ndim, dimf, nInputPlane);
  }

  if (gradOutput != NULL) {
    if (weight != NULL) {
      int64_t nOutputPlane = weight->size(0);
      THCUNN_check_dim_size(state, gradOutput, ndim, dimf, nOutputPlane);
    } else if (bias != NULL) {
      int64_t nOutputPlane = bias->size(0);
      THCUNN_check_dim_size(state, gradOutput, ndim, dimf, nOutputPlane);
    }
    THCUNN_check_dim_size(state, gradOutput, ndim, dimh, outputHeight);
    THCUNN_check_dim_size(state, gradOutput, ndim, dimw, outputWidth);
  }
}

void THNN_(SpatialConvolutionMM_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCTensor *weight,
           THCTensor *bias,
           THCTensor *columns,
           THCTensor *ones,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH) {

  THCUNN_assertSameGPU(state, 5, input, output, weight, columns, ones);
  if (bias) {
    THCUNN_assertSameGPU(state, 2, weight, bias);
  }
  THArgCheck(THCTensor_(isContiguous)(state, weight), 4,
             "weight tensor has to be contiguous");
  THArgCheck(!bias || THCTensor_(isContiguous)(state, bias), 5,
             "bias tensor has to be contiguous");

  int freeWeight = 0;

  // Params:
  int nInputPlane = weight->dim() == 2 ? weight->size(1)/(kH*kW) : weight->size(1);
  int nOutputPlane = weight->size(0);

  if (weight->dim() == 4) {
    int64_t s1 = weight->size(0);
    int64_t s2 = weight->size(1) * weight->size(2) * weight->size(3);
    weight = THCTensor_(newWithStorage2d)(state, THTensor_getStoragePtr(weight), weight->storage_offset(), s1, -1, s2, -1);
    freeWeight = 1;
  }

  THNN_(SpatialConvolutionMM_shapeCheck)
       (state, input, NULL, weight, bias, kH, kW, dH, dW, padH, padW, 0);

  input = THCTensor_(newContiguous)(state, input);
  int is_batch = 1;
  if (input->dim() == 3) {
    // Force batch
    is_batch = 0;
    THCTensor_(resize4d)(state, input, 1, input->size(0), input->size(1), input->size(2));
  }

  int64_t inputWidth   = input->size(3);
  int64_t inputHeight  = input->size(2);
  int64_t outputWidth  = (inputWidth + 2*padW - kW) / dW + 1;
  int64_t outputHeight = (inputHeight + 2*padH - kH) / dH + 1;

  // Batch size + input planes
  int64_t batchSize = input->size(0);

  // Resize output
  THCTensor_(resize4d)(state, output, batchSize, nOutputPlane, outputHeight, outputWidth);

  // Resize temporary columns
  THCTensor_(resize2d)(state, columns, nInputPlane*kW*kH, outputHeight*outputWidth);

  // Define a buffer of ones, for bias accumulation
  // Note: this buffer can be shared with other modules, it only ever gets increased,
  // and always contains ones.
  if (ones->dim() != 2 || ones->size(0)*ones->size(1) < outputHeight*outputWidth) {
    // Resize plane and fill with ones...
    THCTensor_(resize2d)(state, ones, outputHeight, outputWidth);
    THCTensor_(fill)(state, ones, ScalarConvert<int, real>::to(1));
  }

  // Helpers
  THCTensor *input_n = THCTensor_(new)(state);
  THCTensor *output_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCTensor_(select)(state, input_n, input, 0, elt);
    THCTensor_(select)(state, output_n, output, 0, elt);

    // Do Bias first:
    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    int64_t m_ = nOutputPlane;
    int64_t n_ = outputHeight * outputWidth;
    int64_t k_ = 1;

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    if (bias) {
      #ifdef THC_REAL_IS_FLOAT
      THCudaBlas_Sgemm(
      #elif defined(THC_REAL_IS_HALF)
      THCudaBlas_Hgemm(
      #elif defined(THC_REAL_IS_DOUBLE)
      THCudaBlas_Dgemm(
      #endif
          state,
          't', 'n',
          n_, m_, k_,
          ScalarConvert<int, real>::to(1),
          THCTensor_(data)(state, ones), k_,
          THCTensor_(data)(state, bias), k_,
          ScalarConvert<int, real>::to(0),
          THCTensor_(data)(state, output_n), n_
      );
    } else {
      THCTensor_(zero)(state, output_n);
    }

    // Extract columns:
    im2col(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, input_n),
      nInputPlane, inputHeight, inputWidth,
      outputHeight, outputWidth,
      kH, kW, padH, padW, dH, dW,
      1, 1, THCTensor_(data)(state, columns)
    );

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    int64_t m = nOutputPlane;
    int64_t n = columns->size(1);
    int64_t k = nInputPlane*kH*kW;

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #endif
        state,
        'n', 'n',
        n, m, k,
        ScalarConvert<int, real>::to(1),
        THCTensor_(data)(state, columns), n,
        THCTensor_(data)(state, weight), k,
        ScalarConvert<int, real>::to(1),
        THCTensor_(data)(state, output_n), n
    );
  }

  // Free
  THCTensor_(free)(state, input_n);
  THCTensor_(free)(state, output_n);
  if (freeWeight)
    THCTensor_(free)(state, weight);

  // Resize output
  if (is_batch == 0) {
    THCTensor_(resize3d)(state, output, nOutputPlane, outputHeight, outputWidth);
    THCTensor_(resize3d)(state, input, nInputPlane, inputHeight, inputWidth);
  }

  THCTensor_(free)(state, input);
}

void THNN_(SpatialConvolutionMM_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *weight,
           THCTensor *gradColumns,
           THCTensor *ones,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH) {

  THCUNN_assertSameGPU(state, 5, input, gradOutput, weight,
                       gradColumns, gradInput);
  THArgCheck(THCTensor_(isContiguous)(state, weight), 4,
             "weight tensor has to be contiguous");

  THNN_(SpatialConvolutionMM_shapeCheck)
       (state, input, gradOutput, weight, NULL, kH, kW, dH, dW, padH, padW, 0);

  // Params
  int nInputPlane = weight->dim() == 2 ? weight->size(1)/(kW*kH) : weight->size(1);
  int nOutputPlane = weight->size(0);

  int freeWeight = 0;
  if (weight->dim() == 4) {
    int64_t s1 = weight->size(0);
    int64_t s2 = weight->size(1) * weight->size(2) * weight->size(3);
    weight = THCTensor_(newWithStorage2d)(state, THTensor_getStoragePtr(weight), weight->storage_offset(), s1, -1, s2, -1);
    freeWeight = 1;
  }

  input = THCTensor_(newContiguous)(state, input);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  int is_batch = 1;
  if (input->dim() == 3) {
    // Force batch
    is_batch = 0;
    THCTensor_(resize4d)(state, input, 1, input->size(0), input->size(1), input->size(2));
    THCTensor_(resize4d)(state, gradOutput, 1, gradOutput->size(0), gradOutput->size(1), gradOutput->size(2));
  }

  int64_t inputWidth   = input->size(3);
  int64_t inputHeight  = input->size(2);
  int64_t outputWidth  = (inputWidth + 2*padW - kW) / dW + 1;
  int64_t outputHeight = (inputHeight + 2*padH - kH) / dH + 1;

  // Batch size + input planes
  int64_t batchSize = input->size(0);

  // Resize output
  THCTensor_(resize4d)(state, gradInput, batchSize, nInputPlane, inputHeight, inputWidth);

  // Resize temporary columns
  THCTensor_(resize2d)(state, gradColumns, nInputPlane*kW*kH, outputHeight*outputWidth);

  // Helpers
  THCTensor *gradInput_n = THCTensor_(new)(state);
  THCTensor *gradOutput_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per sample:
    THCTensor_(select)(state, gradInput_n, gradInput, 0, elt);
    THCTensor_(select)(state, gradOutput_n, gradOutput, 0, elt);

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    int64_t m = nInputPlane*kW*kH;
    int64_t n = gradColumns->size(1);
    int64_t k = nOutputPlane;

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #endif
        state,
        'n', 't',
        n, m, k,
        ScalarConvert<int, real>::to(1),
        THCTensor_(data)(state, gradOutput_n), n,
        THCTensor_(data)(state, weight), m,
        ScalarConvert<int, real>::to(0),
        THCTensor_(data)(state, gradColumns), n
    );

    // Unpack columns back into input:
    col2im<real, accreal>(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, gradColumns),
      nInputPlane, inputHeight, inputWidth, outputHeight, outputWidth, kH, kW, padH, padW, dH, dW,
      1, 1, THCTensor_(data)(state, gradInput_n)
    );
  }

  // Free
  THCTensor_(free)(state, gradInput_n);
  THCTensor_(free)(state, gradOutput_n);
  if (freeWeight)
    THCTensor_(free)(state, weight);

  // Resize output
  if (is_batch == 0) {
    THCTensor_(resize3d)(state, gradOutput, nOutputPlane, outputHeight, outputWidth);
    THCTensor_(resize3d)(state, input, nInputPlane, inputHeight, inputWidth);
    THCTensor_(resize3d)(state, gradInput, nInputPlane, inputHeight, inputWidth);
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
}

void THNN_(SpatialConvolutionMM_accGradParameters)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradWeight,
           THCTensor *gradBias,
           THCTensor *columns,
           THCTensor *ones,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH,
           accreal scale_) {

  real scale = ScalarConvert<accreal, real>::to(scale_);
  THCUNN_assertSameGPU(state, 5, input, gradOutput, gradWeight, gradBias, columns, ones);
  if (gradWeight) {
    THArgCheck(THCTensor_(isContiguous)(state, gradWeight), 4, "gradWeight needs to be contiguous");
  }
  if (gradBias) {
    THArgCheck(THCTensor_(isContiguous)(state, gradBias), 5, "gradBias needs to be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, ones), 7, "ones needs to be contiguous");
  }

  THNN_(SpatialConvolutionMM_shapeCheck)
       (state, input, gradOutput, gradWeight, gradBias, kH, kW, dH, dW, padH, padW, 1);

  // Params
  input = THCTensor_(newContiguous)(state, input);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  int is_batch = 1;
  if (input->dim() == 3) {
    // Force batch
    is_batch = 0;
    THCTensor_(resize4d)(state, input, 1, input->size(0), input->size(1), input->size(2));
    THCTensor_(resize4d)(state, gradOutput, 1, gradOutput->size(0), gradOutput->size(1), gradOutput->size(2));
  }

  int64_t nInputPlane = input->size(1);
  int64_t nOutputPlane = gradOutput->size(1);

  int freeWeight = 0;
  if (gradWeight && gradWeight->dim() == 4) {
    int64_t s1 = gradWeight->size(0);
    int64_t s2 = gradWeight->size(1) * gradWeight->size(2) * gradWeight->size(3);
    gradWeight = THCTensor_(newWithStorage2d)(state, THTensor_getStoragePtr(gradWeight), gradWeight->storage_offset(), s1, -1, s2, -1);
    freeWeight = 1;
  }

  int64_t inputWidth   = input->size(3);
  int64_t inputHeight  = input->size(2);
  int64_t outputWidth  = (inputWidth + 2*padW - kW) / dW + 1;
  int64_t outputHeight = (inputHeight + 2*padH - kH) / dH + 1;

  // Batch size + input planes
  int64_t batchSize = input->size(0);

  // Define a buffer of ones, for bias accumulation
  if (ones->dim() != 2 || ones->size(0)*ones->size(1) < outputHeight*outputWidth) {
    // Resize plane and fill with ones...
    THCTensor_(resize2d)(state, ones, outputHeight, outputWidth);
    THCTensor_(fill)(state, ones, ScalarConvert<int, real>::to(1));
  }

  // Resize temporary columns
  THCTensor_(resize2d)(state, columns, nInputPlane*kW*kH, outputHeight*outputWidth);

  // Helpers
  THCTensor *input_n = THCTensor_(new)(state);
  THCTensor *gradOutput_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCTensor_(select)(state, gradOutput_n, gradOutput, 0, elt);

    // Do Weight:
    if (gradWeight) {
      // Matrix mulitply per output:
      THCTensor_(select)(state, input_n, input, 0, elt);

      // Extract columns:
      im2col(
        THCState_getCurrentStream(state),
        THCTensor_(data)(state, input_n),
        nInputPlane, inputHeight, inputWidth,
        outputHeight, outputWidth,
        kH, kW, padH, padW, dH, dW,
        1, 1, THCTensor_(data)(state, columns)
      );

      // M,N,K are dims of matrix A and B
      // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
      int64_t m = nOutputPlane;
      int64_t n = nInputPlane*kW*kH;
      int64_t k = columns->size(1);

      // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
      #ifdef THC_REAL_IS_FLOAT
      THCudaBlas_Sgemm(
      #elif defined(THC_REAL_IS_HALF)
      THCudaBlas_Hgemm(
      #elif defined(THC_REAL_IS_DOUBLE)
      THCudaBlas_Dgemm(
      #endif
          state,
          't', 'n',
          n, m, k,
          scale,
          THCTensor_(data)(state, columns), k,
          THCTensor_(data)(state, gradOutput_n), k,
          ScalarConvert<int, real>::to(1),
          THCTensor_(data)(state, gradWeight), n
      );
    }

    // Do Bias:
    if (gradBias) {
      // M,N,K are dims of matrix A and B
      // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
      int64_t m_ = nOutputPlane;
      int64_t k_ = outputHeight * outputWidth;

      // Do GEMV (note: this is a bit confusing because gemv assumes column-major matrices)
      #if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
      #ifdef THC_REAL_IS_FLOAT
      THCudaBlas_Sgemv(
      #elif defined(THC_REAL_IS_DOUBLE)
      THCudaBlas_Dgemv(
      #endif
          state,
          't',
          k_, m_,
          scale,
          THCTensor_(data)(state, gradOutput_n), k_,
          THCTensor_(data)(state, ones), 1,
          ScalarConvert<int, real>::to(1),
          THCTensor_(data)(state, gradBias), 1
      );
      #endif
      #ifdef THC_REAL_IS_HALF
      THCudaBlas_Hgemm(
          state,
          't', 'n',
          m_, 1, k_,
          scale,
          THCTensor_(data)(state, gradOutput_n), k_,
          THCTensor_(data)(state, ones), k_,
          ScalarConvert<int, real>::to(1),
          THCTensor_(data)(state, gradBias), m_
      );
      #endif
    }
  }

  // Free
  THCTensor_(free)(state, input_n);
  THCTensor_(free)(state, gradOutput_n);
  if (freeWeight)
    THCTensor_(free)(state, gradWeight);

  // Resize
  if (is_batch == 0) {
    THCTensor_(resize3d)(state, gradOutput, nOutputPlane, outputHeight, outputWidth);
    THCTensor_(resize3d)(state, input, nInputPlane, inputHeight, inputWidth);
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
}

#endif
